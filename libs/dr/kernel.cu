#include "hip/hip_runtime.h"
#pragma once
#include "kernel.h"

namespace gpu{
  void setup_sim(Sim &sim){
    Mesh & m = *sim.mesh;
    int data_size = (m.node_count * 3);
    double * mass = m.mass.data();
    double * disp = m.displacement.data();
    double * vel = m.velocity.data();
    double * force = m.force.data();
#pragma omp target enter data map(to:mass[0:data_size], disp[0:data_size], vel[0:data_size], force[0:data_size], mass[0:data_size])
  }
  void sync_sim(Sim &sim){
    Mesh & m = *sim.mesh;
    int data_size = (m.node_count * 3);
    double * mass = m.mass.data();
    double * disp = m.displacement.data();
    double * vel = m.velocity.data();
    double * force = m.force.data();
#pragma omp target update from (mass[0:data_size], disp[0:data_size], vel[0:data_size], force[0:data_size], mass[0:data_size])
  }
  void close_sim(Sim &sim){
    Mesh & m = *sim.mesh;
    int data_size = (m.node_count * 3);
    double * mass = m.mass.data();
    double * disp = m.displacement.data();
    double * vel = m.velocity.data();
    double * force = m.force.data();
#pragma omp target exit data map(from:mass[0:data_size], disp[0:data_size], vel[0:data_size], force[0:data_size], mass[0:data_size])
  }
  void update_mps(Sim &sim){
    update_mps_kernel(sim.mps.data(),sim.mesh->displacement.data(),sim.dt,sim.mps.size());
  }
  void update_mps_kernel(mp* mps,double *disp,double dt, int mp_count){
#pragma omp target teams parallel for
    for(int i = 0; i < mp_count;++i){
      mps[i].strain[0] = 0;
    }
  }

  void integrate(Sim &sim){
    auto &m = *sim.mesh;
    integration_kernel(
                       m.mass.data(),
                       m.displacement.data(),
                       m.velocity.data(),
                       m.force.data(),
                       sim.damping_factor,
                       sim.dt,
                      (m.node_count * 3));
  }

  void integration_kernel(double * mass, double * disp, double * vel,double * force,double damping_factor, double dt, int data_size){
  #pragma omp target enter data map(to:mass[0:data_size], disp[0:data_size], vel[0:data_size], force[0:data_size], mass[0:data_size])
  #pragma omp target teams parallel for
    for(int i = 0; i < data_size;++i){
      force[i] += mass[i]*vel[i] * -1.0 * damping_factor;
      vel[i] += force[i]/mass[i] * dt;
      disp[i] += disp[i]/mass[i] * dt;
    }
  #pragma omp target exit data map(from:disp[0:data_size],vel[0:data_size])
  }

  void reset_force(Sim &sim){
    auto &m = *sim.mesh;
    reset_force_kernel(m.force.data(), (m.node_count * 3));
  }
  void reset_force_kernel(double * force, int data_size){
#pragma omp target teams parallel for
    for(int i = 0; i < data_size;++i){
      force[i] = 0.0;
    }
  }


  void apply_bcs(Sim &sim){
    auto &m = *sim.mesh;
    apply_bcs_kernel(m.bcs.data(),
              m.displacement.data(),
              m.velocity.data(),
              m.force.data(),
              (m.node_count * 3));
  }

  void apply_bcs_kernel(double * bcs,double * disp, double *vel,double * force,int data_size){
#pragma omp target enter data map(to:disp[0:data_size], vel[0:data_size], force[0:data_size])
#pragma omp target teams parallel for
    for(int i = 0; i < data_size;++i){
      vel[i] *= bcs[i];
      disp[i] *= bcs[i];
      force[i] *= bcs[i];
    }
#pragma omp target exit data map(from:disp[0:data_size],vel[0:data_size],force[0:data_size])
  }
}

/* Eigen::Matrix<double,6,1> log_strain_update_gpu(const Eigen::Matrix<double,6,1> & strain, const Eigen::Matrix3d& df){ */
/*   Eigen::SelfAdjointEigenSolver<Eigen::Matrix3d> eigensolver(voigt_to_matrix(strain)); */
/*   if (eigensolver.info() != Eigen::Success) */
/*     { */
/*       // std::cout<<"Eigensolve failed\n"; */
/*       // abort(); */
/*       return Eigen::Matrix<double,6,1>::Zero(); */
/*     } */
/*   auto eigen_values = eigensolver.eigenvalues(); */
/*   auto eigen_vectors = eigensolver.eigenvectors(); */
/*   auto trial_lgs = df * (eigen_vectors */
/*                          * (eigen_values.array() * 2.0).exp().matrix().asDiagonal() */
/*                          * eigen_vectors.transpose()) * df.transpose(); */
/*   //0.5 * (trial_lgs + trial_lgs.transpose()) */
/*   Eigen::SelfAdjointEigenSolver<Eigen::Matrix3d> trialeigensolver(trial_lgs); */
/*   if (trialeigensolver.info() != Eigen::Success) */
/*     { */
/*       // std::cout<<"Eigensolve failed\n"; */
/*       // abort(); */
/*       return Eigen::Matrix<double,6,1>::Zero(); */
/*     } */
/*   auto l = trialeigensolver.eigenvalues(); */
/*   auto v = trialeigensolver.eigenvectors(); */
/*   if ((l.array() <= 0.0).any()) */
/*     { */
/*       // std::cout<<"Eigensolve failed\n"; */
/*       // abort(); */
/*       // return false; */
/*       return Eigen::Matrix<double,6,1>::Zero(); */
/*     } */
/*   return (matrix_to_voigt(v * l.array().log().matrix().asDiagonal() * v.transpose()).array() * 0.5).matrix(); */
/* } */

/* void gpu_log_strain_update(mp * mps, int size){ */
/* #pragma omp target teams parallel for */
/*   for(int i = 0; i < size;++i){ */
/*     mps[i].strain = log_strain_update_gpu(mps[i].strain_n,mps[i].df); */
/*   } */
/* } */
