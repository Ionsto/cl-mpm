#include "hip/hip_runtime.h"
#pragma once
#include "kernel.h"

namespace gpu{

  template <typename F>
  void iterate_over_neighbours(mp & mp, F&& func){
    nodecache * nc = mp.nc.data();
#pragma omp target teams parallel for
    for (int i = 0;i < mp.nc.size();++i){
      func(nc[i]);
    }
  }
  void setup_sim(Sim &sim){
    Mesh & m = *sim.mesh;
    int data_size = (m.node_count * 3);
    double * mass = m.mass.data();
    double * disp = m.displacement.data();
    double * vel = m.velocity.data();
    double * force = m.force.data();
    double * bcs = m.bcs.data();
#pragma omp target enter data map(to:mass[0:data_size], disp[0:data_size], vel[0:data_size], force[0:data_size], mass[0:data_size],bcs[0:data_size])
#pragma omp target enter data map(to:sim.mps.data()[sim.mps.size()])
#pragma omp target enter data map(to:sim.mesh->nodes.data()[sim.mesh->nodes.size()])
    for(auto & node : sim.mesh->nodes){
#pragma omp target enter data map(to:node.nc.data()[node.nc.size()])
    }
    for(int index = 0;index < sim.mesh->nodes.size();++index){
      auto & node = sim.mesh->nodes[index];
    }
  }
  void sync_sim(Sim &sim){
    Mesh & m = *sim.mesh;
    int data_size = (m.node_count * 3);
    double * mass = m.mass.data();
    double * disp = m.displacement.data();
    double * vel = m.velocity.data();
    double * force = m.force.data();
#pragma omp target update from (mass[0:data_size], disp[0:data_size], vel[0:data_size], force[0:data_size], mass[0:data_size])
  }
  void close_sim(Sim &sim){
    Mesh & m = *sim.mesh;
    int data_size = (m.node_count * 3);
    double * mass = m.mass.data();
    double * disp = m.displacement.data();
    double * vel = m.velocity.data();
    double * force = m.force.data();
#pragma omp target exit data map(from:mass[0:data_size], disp[0:data_size], vel[0:data_size], force[0:data_size], mass[0:data_size])
  }

  void update_mps(Sim &sim){
    update_mps_kernel(sim.mps.data(),sim.mesh->displacement.data(),sim.dt,sim.mps.size());
  }

  void update_mps_kernel(mp* mps,double *disp,double dt, int mp_count){
#pragma omp target teams parallel for
    for(int i = 0; i < mp_count;++i){
      mps[i].strain[0] = 0;
    }
  }

  void integrate(Sim &sim){
    auto &m = *sim.mesh;
    integration_kernel(m.mass.data(),
                       m.displacement.data(),
                       m.velocity.data(),
                       m.force.data(),
                       sim.damping_factor,
                       sim.dt,
                       (m.node_count * 3));
  }

  void integration_kernel(double * mass, double * disp, double * vel,double * force,double damping_factor, double dt, int data_size){
  // #pragma omp target enter data map(to:mass[0:data_size], disp[0:data_size], vel[0:data_size], force[0:data_size], mass[0:data_size])
#pragma omp target update from (mass[0:data_size], disp[0:data_size], vel[0:data_size], force[0:data_size])
  #pragma omp target teams parallel for
    for(int i = 0; i < data_size;++i){
      force[i] += mass[i]*vel[i] * -1.0 * damping_factor;
      vel[i] += force[i]/mass[i] * dt;
      disp[i] += disp[i]/mass[i] * dt;
    }
  // #pragma omp target exit data map(from:disp[0:data_size],vel[0:data_size])
  }

  void reset_force(Sim &sim){
    auto &m = *sim.mesh;
    reset_force_kernel(m.force.data(), (m.node_count * 3));
  }
  void reset_force_kernel(double * force, int data_size){
#pragma omp target teams distribute parallel for simd
    for(int i = 0; i < data_size;++i){
      force[i] = 0.0;
    }
  }


  void apply_bcs(Sim &sim){
    auto &m = *sim.mesh;
    apply_bcs_kernel(m.bcs.data(),
              m.displacement.data(),
              m.velocity.data(),
              m.force.data(),
              (m.node_count * 3));
  }

  void apply_bcs_kernel(double * bcs,double * disp, double *vel,double * force,int data_size){
#pragma omp target update from (bcs[0:data_size], disp[0:data_size], vel[0:data_size], force[0:data_size])
#pragma omp target teams parallel for
    for(int i = 0; i < data_size;++i){
      vel[i] *= bcs[i];
      disp[i] *= bcs[i];
      force[i] *= bcs[i];
    }
  }

  Dsvp assemble_dsvp(Vector grads){
    Dsvp dsvp_alloc = Dsvp::Zero();
    dsvp_alloc(0,0) = grads[0];
    dsvp_alloc(1,1) = grads[1];
    dsvp_alloc(2,2) = grads[2];

    dsvp_alloc(3,1) = grads[2];
    dsvp_alloc(3,2) = grads[1];

    dsvp_alloc(4,0) = grads[2];
    dsvp_alloc(4,2) = grads[0];

    dsvp_alloc(5,0) = grads[1];
    dsvp_alloc(5,1) = grads[0];
    return dsvp_alloc;
  }

  void p2g_gather_force(Sim& sim){
    p2g_gather_force_kernel(sim.mesh->nodes.data(),sim.mps.data(),sim.mesh->force.data(),sim.mesh->nodes.size());
  }
  void p2g_gather_force_kernel(node * nodes, mp * mps,double * force,int node_count)
  {
#pragma omp target teams parallel for
    for (int i = 0;i < node_count;++i){
      nodecache * nc = nodes[i].nc.data();
      force[nc[0].node*3]=1;
      // for(int j = 0;j < nodes[i].nc.size();++j){
        // func(nc[i]);

        // mps[nc[j].mp].stress[0] = 1.0;
        // Dsvp dsvp = assemble_dsvp(mp.df.inverse()*nc[i].grads);
        // force[i*3] = 1.0;
        // Eigen::Vector<double,1,3> force_row(&force[i*3]);
        // force_row += (-1.0*mp.volume*(dsvp.transpose() * mp.stress).transpose()) + (mp.mass * n.svp * sim.mesh->gravity.transpose());
      // }
    }
  }
}

/* Eigen::Matrix<double,6,1> log_strain_update_gpu(const Eigen::Matrix<double,6,1> & strain, const Eigen::Matrix3d& df){ */
/*   Eigen::SelfAdjointEigenSolver<Eigen::Matrix3d> eigensolver(voigt_to_matrix(strain)); */
/*   if (eigensolver.info() != Eigen::Success) */
/*     { */
/*       // std::cout<<"Eigensolve failed\n"; */
/*       // abort(); */
/*       return Eigen::Matrix<double,6,1>::Zero(); */
/*     } */
/*   auto eigen_values = eigensolver.eigenvalues(); */
/*   auto eigen_vectors = eigensolver.eigenvectors(); */
/*   auto trial_lgs = df * (eigen_vectors */
/*                          * (eigen_values.array() * 2.0).exp().matrix().asDiagonal() */
/*                          * eigen_vectors.transpose()) * df.transpose(); */
/*   //0.5 * (trial_lgs + trial_lgs.transpose()) */
/*   Eigen::SelfAdjointEigenSolver<Eigen::Matrix3d> trialeigensolver(trial_lgs); */
/*   if (trialeigensolver.info() != Eigen::Success) */
/*     { */
/*       // std::cout<<"Eigensolve failed\n"; */
/*       // abort(); */
/*       return Eigen::Matrix<double,6,1>::Zero(); */
/*     } */
/*   auto l = trialeigensolver.eigenvalues(); */
/*   auto v = trialeigensolver.eigenvectors(); */
/*   if ((l.array() <= 0.0).any()) */
/*     { */
/*       // std::cout<<"Eigensolve failed\n"; */
/*       // abort(); */
/*       // return false; */
/*       return Eigen::Matrix<double,6,1>::Zero(); */
/*     } */
/*   return (matrix_to_voigt(v * l.array().log().matrix().asDiagonal() * v.transpose()).array() * 0.5).matrix(); */
/* } */

/* void gpu_log_strain_update(mp * mps, int size){ */
/* #pragma omp target teams parallel for */
/*   for(int i = 0; i < size;++i){ */
/*     mps[i].strain = log_strain_update_gpu(mps[i].strain_n,mps[i].df); */
/*   } */
/* } */
