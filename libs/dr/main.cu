#include "hip/hip_runtime.h"
#include <iostream>
#include <array>
#include <vector>
#include <chrono>
#include <Eigen/Dense>
// #include "types.h"
// #include "hot.cuh"


void integrate_kernel(float *x,float* y,float* z,int size){
#pragma omp target teams parallel for
  for(int i = 0;i < size;++i)
    {
      z[i] = x[i] + y[i];
    }
}

// void add(mp * mps,int size){
// // #pragma omp target map(tofrom:mps[0,size])
// #pragma omp target teams parallel for
//   for(int i = 0;i < size;++i)
//     {
//       mps[i].stress = mps[i].de *  mps[i].strain;
//     }
// }

int main(int argc, char **args){
  typedef std::chrono::high_resolution_clock Clock;
  typedef Eigen::Matrix<float,Eigen::Dynamic,3> DofType;
  const int data_size = 10000;
  const int iters = 100;
  const int total_iters = iters;
  // std::vector<mp> x(data_size);

  DofType x = DofType::Zero(data_size,3);
  DofType y = DofType::Zero(data_size,3);
  DofType z = DofType::Zero(data_size,3);

  std::cout << "Mps: "<< data_size<<"\n";
  std::cout << "Iters: "<< iters <<"\n";
#pragma omp target enter data map(to:x.data()[0:data_size],y.data()[0:data_size])
  auto t1 = Clock::now();
  for(int i = 0;i < iters;++i){
    integrate_kernel(x.data(),y.data(),z.data(),data_size);
    // add(x.data(),data_size);
  }
  auto t2 = Clock::now();
#pragma omp target exit data map(from:z.data()[0:data_size])
  // int print_size = std::min(3,data_size);
  // for(int i = 0;i < print_size;++i){
  //   std::cout<<x[i].strain[0]<<"\n";
  // }
  std::cout << "Took: " << (std::chrono::duration_cast<std::chrono::duration<double>>(t2-t1)).count() << " seconds \n";
  std::cout << "Throughput: " << total_iters / (std::chrono::duration_cast<std::chrono::duration<double>>(t2-t1)).count() << " GFLOPs \n";

}
